
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"


#define MASSA 1
#define EPSILON 1E-9

typedef struct vetor
{
    double x, y, z;
} VETOR;

typedef struct posicao
{
    double x, y, z;
} POSICAO;

typedef struct particula
{
    POSICAO coord;
    VETOR forca_sofrida;
    VETOR velocidade;
} PARTICULA;

void inicializador(PARTICULA *particula, int quantidade)
{
    srand(42);
    memset(particula, 0x00, quantidade * sizeof(PARTICULA));
    for (int i = 0; i < quantidade; i++)
    {
        particula[i].coord.x = 2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particula[i].coord.y = 2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particula[i].coord.z = 2.0 * (rand() / (double)RAND_MAX) - 1.0;
    }
}

void printLog(PARTICULA *particles, int quantParticulas, int timestep, char *type)
{
    char path[100] = "../";
    sprintf(path, "Log/%s/Log%d-log.txt", type, timestep);
    fprintf(stdout, "Saving file [%s] ", path);
    fflush(stdout);
    FILE *arquivo = fopen(path, "w+");
    for (int i = 0; i < quantParticulas; i++)
    {
        fprintf(arquivo, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n",
                i,
                particles[i].coord.x, particles[i].coord.y, particles[i].coord.z,
                particles[i].velocidade.x, particles[i].velocidade.y, particles[i].velocidade.z,
                particles[i].forca_sofrida.x, particles[i].forca_sofrida.y, particles[i].forca_sofrida.z);
    }
    fclose(arquivo);
    fprintf(stdout, "[OK]\n");
    fflush(stdout);
}

__device__ 
void atualizaVelocidade(PARTICULA *particula, double dt)
{
    particula->velocidade.x += dt * particula->forca_sofrida.x;
    particula->velocidade.y += dt * particula->forca_sofrida.y;
    particula->velocidade.z += dt * particula->forca_sofrida.z;
}

__device__ 
void atualizaCoordenada(PARTICULA *particula, double dt)
{
    particula->coord.x += dt * particula->velocidade.x;
    particula->coord.y += dt * particula->velocidade.y;
    particula->coord.z += dt * particula->velocidade.z;
}

__global__ 
void simulacao(PARTICULA *particula, int quantParticulas, double dt)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < quantParticulas)
    {

        for (int j = 0; j < quantParticulas; j++)
        {
            if (i != j)
            {
                double dx = particula[i].coord.x - particula[j].coord.x;
                double dy = particula[i].coord.y - particula[j].coord.y;
                double dz = particula[i].coord.z - particula[j].coord.z;
                double dist = dx * dx + dy * dy + dz * dz + EPSILON;
                double distSqrRoot = sqrt(dist);
                double invDist = 1.0 / pow(distSqrRoot, 2);

                particula[i].forca_sofrida.x += dx * invDist;
                particula[i].forca_sofrida.y += dy * invDist;
                particula[i].forca_sofrida.z += dz * invDist;
            }
        }
        atualizaVelocidade(&particula[i], dt);
        __syncthreads();

        atualizaCoordenada(&particula[i], dt);
        __syncthreads();
    }
}

int main(int ac, char **av)
{
    int timesteps = atoi(av[1]), quantParticulas = atoi(av[2]), flagSave = atoi(av[3]);

    clock_t t;
    t = clock();

    char logFile[1024];
    double dt = 0.01;
    PARTICULA *particulas = NULL;
    PARTICULA *d_particula;

    strcpy(logFile, av[4]);

    fprintf(stdout, "\nSistema de particulas P2P(particula a particula)\n");
    fprintf(stdout, "Memória utilizada %lu bytes \n", quantParticulas * sizeof(PARTICULA));
    fprintf(stdout, "Arquivo %s \n", logFile);

    particulas = (PARTICULA *)malloc(quantParticulas * sizeof(PARTICULA));
    assert(particulas != NULL);

    inicializador(particulas, quantParticulas);

    int block_size = 1024;
    int grid_size = ((quantParticulas + block_size - 1) / block_size);

    hipMalloc(&d_particula, sizeof(PARTICULA) * quantParticulas);

    hipMemcpy(d_particula, particulas, sizeof(PARTICULA) * quantParticulas, hipMemcpyHostToDevice);

    for (int j = 0; j < timesteps; j++)
    {
        simulacao<<<grid_size, block_size>>>(d_particula, quantParticulas, dt);
    }

    hipMemcpy(particulas, d_particula, sizeof(PARTICULA) * quantParticulas, hipMemcpyDeviceToHost);

    hipFree(d_particula);

    t = clock() - t;
    double time_taken = ((double)t) / CLOCKS_PER_SEC;
    fprintf(stdout, "Tempo gasto: %lf (s) \n\n", time_taken);

    FILE *log = fopen(logFile, "a+");
    assert(log != NULL);
    fprintf(log, "Timesteps: %d\nNúmero de Particulas: %d\nMemória em bytes:%lu\nTempo em segundos:%lf\n-----------------------------\n", timesteps, quantParticulas, quantParticulas * sizeof(particulas), time_taken);
    fclose(log);

    if (flagSave == 1)
        printLog(particulas, quantParticulas, timesteps, "ParallelCuda");
    free(particulas);
}
